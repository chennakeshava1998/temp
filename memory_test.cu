#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <math.h>

// Number of rows in the first input matrix
#define M (int)(pow(2, 32))

// Number of columns in the first input matrix
#define N (int)(pow(2, 32))
int cnt;
void have_fun()
{
    // to prevent a seg-fault
    if(cnt < 130080)
    return;

    // printf("Number of function calls: %d\n", cnt);
    ++cnt;
    double *A, *B, *C;
    hipMalloc((void **)&A, M * N * sizeof(double));
    hipMalloc((void **)&B, M * N * sizeof(double));
    hipMalloc((void **)&C, M * M * sizeof(double));


    hipMemcpy(A, host_A, M * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(B, host_B, M * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(host_C, C, M * M * sizeof(double), hipMemcpyDeviceToHost);
    

    have_fun();
    --cnt;
    hipFree(A);
    hipFree(B);
    hipFree(C);
}

int main()
{
    cnt = 0;
    int i = 0;
    while(1)
    {
        printf("i = %d\n", i);
        ++i;
        have_fun();
    }
        

    return 0;
}
