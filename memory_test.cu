#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <math.h>

// Number of rows in the first input matrix
#define M (int)(pow(2, 32))

// Number of columns in the first input matrix
#define N (int)(pow(2, 32))
int cnt;
void have_fun()
{
    // printf("Number of function calls: %d\n", cnt);
    ++cnt;
    double *A, *B, *C;
    hipMalloc((void **)&A, M * N * sizeof(double));
    hipMalloc((void **)&B, M * N * sizeof(double));
    hipMalloc((void **)&C, M * M * sizeof(double));

    have_fun();
}



int main()
{
    cnt = 0;
    have_fun();

    return 0;
}
